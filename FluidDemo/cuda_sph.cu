#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <cuda_D3D11_interop.h>

#include <stdio.h>
#include <iostream>
#include "MyMath.h"
#include "MyVector3.h"

#include <D3D11.h>

struct hipGraphicsResource* positionsVB_CUDA;

//-------------GPU�õ�����--------------------
__constant__ static const  float  unitScale = 0.012f;			// �ߴ絥λ
__constant__ static const float unitScale2 = 0.000144f;		//�߶ȵ�λƽ��
__constant__ static const float viscosity = 1.0f;				// ճ��
__constant__ static const float restDensity = 1000.f;			// �ܶ�
__constant__ static const float pointMass = 0.0004f;			// ��������
//__constant__ static const float gasConstantK = 1.0f;				// �������巽�̳���
__constant__ static const float  gpu_smoothRadius = 0.01f;			// �⻬�˰뾶

__constant__ static const float boundartStiffness = 10000.f;
__constant__ static const float boundaryDampening = 256.f;
__constant__ static const float speedLimiting = 200.f;

__constant__ static const   float edgeWidth = 2.f;//�߽���

__constant__ static const  float gravityY = -9.8f;

__constant__ static const  float deltaTime = 0.003f;

//__constant__ static const  int gpu_particleCount = 10000;

__constant__ static const  int gpu_maxParticlePerGrid = 100;

__constant__ static const  int gpu_maxNeighborCount = 60;

//-------------CPU�õ�����-----------------
const int blockCount = 20;

const int particleCount = 10000;

const int maxParticlePerGrid = 100;

const int maxNeighborCount = 60;

const int	W = 15;

const int H = 15;

const int GRIDH = 30;

const float scale = 0.012f;
const float smoothRadius = 0.01f;			// �⻬�˰뾶
float cellSize = 2 * smoothRadius;

const float cpuDensity = 1000.f;			// �ܶ�
const float cpuMass = 0.0004f;			// ��������

//Poly6 Kernel
float kernelPoly6 = 315.0f / (64.0f * 3.141592f * pow(smoothRadius, 9));
//Spiky Kernel
float kernelSpiky = -45.0f / (3.141592f * pow(smoothRadius, 6));
//Viscosity Kernel
float kernelViscosity = 45.0f / (3.141592f * pow(smoothRadius, 6));


struct Box
{
	Box() = default;
	Box(Box& box) { mMin = box.mMin; mDim = box.mDim; }
	void operator=(Box& box){ mMin = box.mMin; mDim = box.mDim; }
	MyMath::float3 mMin{ 0.f, 0.f, 0.f };
	MyMath::int3 mDim;// { 0, 0, 0 };//��������Max���������ֵ��������2h��
};

struct WaterBox
{
	float minX;
	float minZ;
	float minY;
	float maxX;
	float maxZ;
};

struct Particle
{
	MyMath::float3 mPos{ 0, 0, 0 };
	MyMath::float3 mVec{ 0, 0, 0 };
	MyMath::float3	 mAccel{ 0, 0, 0 };	//���ٶ�

	MyMath::float3		mVecEval{ 0, 0, 0 };

	float			mDensity{ 0 };	//�ܶ�
	float			mPressure{ 0 };	//ѹ��
};

struct hashUnit
{
	int mGridIndex{ 0 };
	int mParticleIndex{ 0 };
};

struct indexUnit
{
	int mBegin{ 0 };
	int mEnd{ 0 };
};

__device__ void swap(hashUnit*const hashTable, int i, int j)
{
	hashUnit temp = hashTable[i];
	hashTable[i] = hashTable[j];
	hashTable[j] = temp;
}

__global__ void zeroGridCount(int*const orderedGridContainer)
{
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x + ((blockIdx.y*blockDim.y) + threadIdx.y)*blockDim.x*gridDim.x;

	orderedGridContainer[tid * gpu_maxParticlePerGrid] = 0;
}

__global__ void buidGridContent(Particle*const container, Box*const gridBox, int*const orderedGridContainer)
{
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x + ((blockIdx.y*blockDim.y) + threadIdx.y)*blockDim.x*gridDim.x;

	const  float cellSize = gpu_smoothRadius * 2 / unitScale;
	MyMath::float3 pos = container[tid].mPos;
	MyMath::float3 min = gridBox->mMin;
	MyMath::int3 dim = gridBox->mDim;

	//����ÿ��cell�����ӵ�������
	int index = ((int)((pos.y - min.y) / cellSize)*dim.x*dim.z + (int)((pos.z - min.z) /
		cellSize)*dim.x + (int)((pos.x - min.x) / cellSize)) * gpu_maxParticlePerGrid;
	int offset = atomicAdd(&orderedGridContainer[index], 1);
	offset += 1;
	orderedGridContainer[index + offset] = tid;
}

__device__ void findCells(int*cells, Particle& particle, Box* box)
{
	for (int i = 0; i < 8; i++)
	{
		cells[i] = -1;
	}

	float r = gpu_smoothRadius / unitScale;
	// ������ķ�Χ(grid)
	int sph_min_x = (int)((-r + particle.mPos.x - box->mMin.x) / 2 / r);
	int sph_min_y = (int)((-r + particle.mPos.y - box->mMin.y) / 2 / r);
	int sph_min_z = (int)((-r + particle.mPos.z - box->mMin.z) / 2 / r);
	if (sph_min_x < 0) sph_min_x = 0;
	if (sph_min_y < 0) sph_min_y = 0;
	if (sph_min_z < 0) sph_min_z = 0;

	cells[0] = (sph_min_y*box->mDim.z + sph_min_z)*box->mDim.x + sph_min_x;
	cells[1] = cells[0] + 1;
	cells[2] = (int)(cells[0] + box->mDim.x);
	cells[3] = cells[2] + 1;

	if (sph_min_y + 1 < box->mDim.y)
	{
		cells[4] = (int)(cells[0] + box->mDim.x*box->mDim.z);
		cells[5] = cells[4] + 1;
		cells[6] = (int)(cells[4] + box->mDim.x);
		cells[7] = cells[6] + 1;
	}
	if (sph_min_x + 1 >= box->mDim.x)
	{
		cells[1] = -1;		cells[3] = -1;
		cells[5] = -1;		cells[7] = -1;
	}
	if (sph_min_z + 1 >= box->mDim.z)
	{
		cells[2] = -1;		cells[3] = -1;
		cells[6] = -1;		cells[7] = -1;
	}
}

__device__ float square(MyMath::float3& p0, MyMath::float3& p1)
{
	return pow(p0.x - p1.x, 2) + pow(p0.y - p1.y, 2) + pow(p0.z - p1.z, 2);
}

__device__ float length(MyMath::float3& p0, MyMath::float3& p1)
{
	return sqrt(pow(p0.x - p1.x, 2) + pow(p0.y - p1.y, 2) + pow(p0.z - p1.z, 2));
}

__device__ float lengthSqr(MyMath::float3& p)
{
	return p.x*p.x + p.y*p.y + p.z*p.z;
}

__device__ float evolution(float input)
{
	return sqrt(input);
}

__global__ void computeDensityAndPressure(int*const orderedGridContainer, Particle*const container, Box*const box, float*const kernelPoly6, int*const neighborTable)
{
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x + ((blockIdx.y*blockDim.y) + threadIdx.y)*blockDim.x*gridDim.x;

	float sum = 0;
	float h2 = gpu_smoothRadius*gpu_smoothRadius;
	//int maxNeighborCount = gpu_maxNeighborCount
	//�ҵ��̶߳�Ӧ������
	Particle selfParticle = container[tid];
	//8�������ڹ⻬���Ӱ뾶�ڵ�cells
	int cells[8];
	findCells(cells, selfParticle, box);

	int neighborCount = 0;

	//ͨ������indexTable��ÿ��cell�е����ӽ��б���
	for (int i = 0; i < 8; i++)
	{
		if (~cells[i])
		{
			int index = cells[i] * gpu_maxParticlePerGrid;
			//int num = orderedGridContainer[index];

			for (int j = 1; j <= orderedGridContainer[index]; ++j)
			{
				int neighborIndex = orderedGridContainer[index + j];
				if (tid == neighborIndex)
				{
					sum +=h2 * h2 * h2;
				}
				else
				{
					MyMath::float3 neighborPos = container[neighborIndex].mPos;
					float disX = selfParticle.mPos.x - neighborPos.x;
					float disY = selfParticle.mPos.y - neighborPos.y;
					float disZ = selfParticle.mPos.z - neighborPos.z;
					float dis2 = (disX*disX + disY*disY + disZ*disZ)*unitScale2;//*unitScale;

					//����������С�ڹ⻬���Ӱ뾶
					if (dis2 < h2)
					{
						++neighborCount;
						//���м���
						sum += (h2 - dis2) * (h2 - dis2) * (h2 - dis2);
						int neiborIndex = tid * gpu_maxNeighborCount + neighborCount;
						neighborTable[neiborIndex] = neighborIndex;
						//dis2Table[neiborIndex] = dis2;
						if (neighborCount == gpu_maxNeighborCount - 1)
						{
							i = 8;
							break;
						}
					}
				}
			}
		}
	}

	neighborTable[tid * gpu_maxNeighborCount] = neighborCount;

	//�����ܶȺ�ѹ��
	container[tid].mDensity = (*kernelPoly6)*pointMass*sum;
	float temp = container[tid].mDensity / restDensity;
	container[tid].mPressure = 100.f*(temp*temp*temp*temp*temp*temp - 1);//gasConstantK*(container[tid].mDensity - restDensity);
}

__global__ void computeForce(Particle*const container, float*const kernelSpiky, float*const kernelViscosity, int*const neighborTable)
{
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x + ((blockIdx.y*blockDim.y) + threadIdx.y)*blockDim.x*gridDim.x;

	MyMath::float3 sum0;
	MyMath::float3 sum1;

	int neiborBeginPos = tid * gpu_maxNeighborCount;
	//�ҵ��̶߳�Ӧ������
	Particle tempParticle = container[tid];
	int temp = neighborTable[neiborBeginPos];
	//Particle neighborParticle = container[neighborTable[neiborBeginPos + j]];
	//ͨ������neighborTable��ÿ���ڽ����ӽ��б���
	for (int j = 1; j <= temp; ++j)
	{
		Particle neighborParticle = container[neighborTable[neiborBeginPos + j]];
		//float dis = length(neighborParticle.mPos*unitScale, tempParticle.mPos*unitScale);//�����Ӿ���
		//float dis = sqrt(dis2Table[neiborBeginPos + j]);
		float disX = tempParticle.mPos.x - neighborParticle.mPos.x;
		float disY = tempParticle.mPos.y - neighborParticle.mPos.y;
		float disZ = tempParticle.mPos.z - neighborParticle.mPos.z;
		float dis = sqrt(disX*disX + disY*disY + disZ*disZ)*unitScale;

		float h_r = gpu_smoothRadius - dis;
		//���м���
		sum0 += (tempParticle.mPos - neighborParticle.mPos) *((tempParticle.mPressure + neighborParticle.mPressure) / neighborParticle.mDensity*h_r*h_r / dis);

		sum1 += (neighborParticle.mVecEval - tempParticle.mVecEval) *(h_r / neighborParticle.mDensity);
	}

	//������������õļ��ٶ�
	container[tid].mAccel = sum0*((-*kernelSpiky)*unitScale / (2 * tempParticle.mDensity)*pointMass) + sum1*(viscosity**kernelViscosity / tempParticle.mDensity*pointMass);
}

__device__ float dot(MyMath::float3& para0, MyMath::float3& para1)
{
	return para0.x*para1.x + para0.y*para1.y + para0.z*para1.z;
}

__global__ void advance(Particle*const container, WaterBox*const box)
{
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x + ((blockIdx.y*blockDim.y) + threadIdx.y)*blockDim.x*gridDim.x;

	Particle particle = container[tid];
	MyMath::float3 acc = particle.mAccel;

	//����
	MyMath::float3 g(0.f, gravityY, 0.f);
	acc += g;

	//�ٶ�����
	float sl2 = speedLimiting*speedLimiting;
	float acc2 = acc.x*acc.x + acc.y*acc.y + acc.z*acc.z;//lengthSqr(acc);
	if (acc2>sl2)
	{
		acc = acc*speedLimiting / sqrt(acc2);
	}

	// �߽�����
	//Z����
	float diff = (edgeWidth - (particle.mPos.z - box->minZ))*unitScale;
	if (diff>0)
	{
		MyMath::float3 normal(0.f, 0.f, 1.f);
		float adj = boundartStiffness * diff - boundaryDampening *(normal.x*particle.mVecEval.x + normal.y*particle.mVecEval.y + normal.z*particle.mVecEval.z);//dot(normal, particle.mVecEval);
		acc.x += adj * normal.x;
		acc.y += adj * normal.y;
		acc.z += adj * normal.z;
	}

	diff = (edgeWidth - (box->maxZ - particle.mPos.z))*unitScale;
	if (diff>0)
	{
		MyMath::float3 normal(0.f, 0.f, -1.f);
		float adj = boundartStiffness * diff - boundaryDampening * (normal.x*particle.mVecEval.x + normal.y*particle.mVecEval.y + normal.z*particle.mVecEval.z);
		acc.x += adj * normal.x;
		acc.y += adj * normal.y;
		acc.z += adj * normal.z;
	}

	//X����
	diff = (edgeWidth - (particle.mPos.x - box->minX))*unitScale;
	if (diff>0)
	{
		MyMath::float3 normal(1.f, 0.f, 0.f);
		float adj = boundartStiffness * diff - boundaryDampening * (normal.x*particle.mVecEval.x + normal.y*particle.mVecEval.y + normal.z*particle.mVecEval.z);
		acc.x += adj * normal.x;
		acc.y += adj * normal.y;
		acc.z += adj * normal.z;
	}

	diff = (edgeWidth - (box->maxX - particle.mPos.x))*unitScale;
	if (diff>0)
	{
		MyMath::float3 normal(-1.f, 0.f, 0.f);
		float adj = boundartStiffness * diff - boundaryDampening * (normal.x*particle.mVecEval.x + normal.y*particle.mVecEval.y + normal.z*particle.mVecEval.z);
		acc.x += adj * normal.x;
		acc.y += adj * normal.y;
		acc.z += adj * normal.z;
	}

	//Y����
	diff = (edgeWidth - (particle.mPos.y - box->minY))*unitScale;
	if (diff>0)
	{
		MyMath::float3 normal(0.f, 1.f, 0.f);
		float adj = boundartStiffness * diff - boundaryDampening * (normal.x*particle.mVecEval.x + normal.y*particle.mVecEval.y + normal.z*particle.mVecEval.z);
		acc.x += adj * normal.x;
		acc.y += adj * normal.y;
		acc.z += adj * normal.z;
	}




	//�����ٶȺ�λ��
	MyMath::float3 vNext = particle.mVec + acc*deltaTime;
	container[tid].mVecEval = (particle.mVec + vNext)*0.5f;
	container[tid].mVec = vNext;
	container[tid].mPos = particle.mPos + vNext*deltaTime/unitScale;

}

WaterBox waterBox;
WaterBox* gpu_waterBox;

Box* gpu_gridBox;
Box cpu_gridBox;

//hashUnit cpu_hashTable[particleCount];
//hashUnit* gpu_hashTable;

Particle cpu_container[particleCount];
Particle* gpu_container;

int* gpu_orderedGridContainer;

int cpu_gridCount;
int* gpu_gridCount;

float* gpu_poly6;
float* gpu_kernelSpiky;
float* gpu_kernelViscosity;

int* gpu_neighborTable;
//float* gpu_dis2Table;

void resetWaterBox()
{
	waterBox.minX = 0.f;
	waterBox.minZ = 0.f;
	waterBox.minY = 0.f;
	waterBox.maxX = W * (cellSize / scale);
	waterBox.maxZ = H * (cellSize / scale);
}

void initeWaterPos()
{
	float pointDistance = pow(cpuMass / cpuDensity, 1.f / 3.f) / scale; //���Ӽ��
	MyMath::int3 fuidDim;
	fuidDim.x = (int)(cpu_gridBox.mDim.x / 2 * cellSize / scale / pointDistance);
	fuidDim.z = (int)(cpu_gridBox.mDim.z / 2 * cellSize / scale / pointDistance);
	fuidDim.y = 0;
	//��ʼ��container
	const int baseHeight = 5;
	int id = 0;
	bool FULL = false;
	while (true)
	{
		for (int z = 0; z < fuidDim.z; ++z)
		{
			for (int x = 0; x < fuidDim.x; ++x)
			{
				cpu_container[id].mPos.x = (waterBox.maxX - waterBox.minX) / 4 + x*pointDistance;
				cpu_container[id].mPos.z = (waterBox.maxZ - waterBox.minZ) / 4 + z*pointDistance;
				cpu_container[id].mPos.y = baseHeight + fuidDim.y*pointDistance;
				++id;
				if (id == particleCount)
				{
					FULL = true;
					break;
				}
			}
			if (FULL)
			{
				break;
			}
		}
		if (FULL)
		{
			break;
		}
		++fuidDim.y;
	}
}

//��ʼ������
extern "C" void initData()
{
	//����ˮ��
	resetWaterBox();

	int dimX = (int)((waterBox.maxX - waterBox.minX) / (cellSize / scale));
	if (((waterBox.maxX - waterBox.minX) / (cellSize / scale))>dimX)
	{
		dimX += 1;
	}
	int dimZ = (int)((waterBox.maxZ - waterBox.minZ) / (cellSize / scale));
	if (((waterBox.maxZ - waterBox.minZ) / (cellSize / scale))>dimZ)
	{
		dimZ += 1;
	}
	hipMalloc((void**)&gpu_waterBox, sizeof(WaterBox));
	hipMemcpy(gpu_waterBox, &waterBox, sizeof(WaterBox), hipMemcpyHostToDevice);

	//��ʼ��gridBox
	cpu_gridBox.mMin = { waterBox.minX, waterBox.minY, waterBox.minZ };
	cpu_gridBox.mDim = { dimX, 30, dimZ };
	hipMalloc((void**)&gpu_gridBox, sizeof(Box));
	hipMemcpy(gpu_gridBox, &cpu_gridBox, sizeof(Box), hipMemcpyHostToDevice);

	//��GPU�ϸ�hashTable�����ڴ�
	//hipMalloc((void**)&gpu_hashTable, sizeof(hashUnit)*particleCount);

	//��ʼ��ˮ��λ��
	initeWaterPos();

	hipMalloc((void**)&gpu_container, sizeof(Particle)*particleCount);
	hipMemcpy(gpu_container, cpu_container, sizeof(Particle)*particleCount, hipMemcpyHostToDevice);

	hipMalloc((void**)&gpu_orderedGridContainer, sizeof(int)* maxParticlePerGrid * particleCount);

	//��GPU����������
	cpu_gridCount = cpu_gridBox.mDim.x*cpu_gridBox.mDim.y*cpu_gridBox.mDim.z;
	hipMalloc((void**)&gpu_gridCount, sizeof(int));
	hipMemcpy(gpu_gridCount, &cpu_gridCount, sizeof(int), hipMemcpyHostToDevice);

	//��GPU��N-S kernelֵ
	hipMalloc((void**)&gpu_poly6, sizeof(float));
	hipMemcpy(gpu_poly6, &kernelPoly6, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&gpu_kernelSpiky, sizeof(float));
	hipMemcpy(gpu_kernelSpiky, &kernelSpiky, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&gpu_kernelViscosity, sizeof(float));
	hipMemcpy(gpu_kernelViscosity, &kernelViscosity, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&gpu_neighborTable, sizeof(int)*particleCount * maxNeighborCount);
	
	//hipMalloc((void**)&gpu_dis2Table, sizeof(int)*particleCount * maxNeighborCount);
}

//�ͷ���Դ
extern "C" void shutDown()
{
	hipFree(gpu_gridBox);
	hipFree(gpu_container);
	//hipFree(gpu_hashTable);
	hipFree(gpu_gridCount);
	hipFree(gpu_poly6);
	hipFree(gpu_kernelSpiky);
	hipFree(gpu_kernelViscosity);
	hipFree(gpu_neighborTable);
	//hipFree(gpu_dis2Table);
	hipFree(gpu_orderedGridContainer);
	hipFree(gpu_waterBox);

}

bool boxIsChange = false;
bool reset = false;
void updateBox()
{
	if (reset)
	{
		resetWaterBox();
	}

	int dimX = (int)((waterBox.maxX - waterBox.minX) / (cellSize / scale));
	if (((waterBox.maxX - waterBox.minX) / (cellSize / scale)) > dimX)
	{
		dimX += 1;
	}
	int dimZ = (int)((waterBox.maxZ - waterBox.minZ) / (cellSize / scale));
	if (((waterBox.maxZ - waterBox.minZ) / (cellSize / scale)) > dimZ)
	{
		dimZ += 1;
	}
	cpu_gridBox.mMin = { waterBox.minX, waterBox.minY, waterBox.minZ };
	cpu_gridBox.mDim = { dimX, GRIDH, dimZ };

	hipMemcpy(gpu_waterBox, &waterBox, sizeof(WaterBox), hipMemcpyHostToDevice);
	hipMemcpy(gpu_gridBox, &cpu_gridBox, sizeof(Box), hipMemcpyHostToDevice);

	boxIsChange = false;

	if (reset)
	{
		initeWaterPos();
		hipMemcpy(gpu_container, cpu_container, sizeof(Particle)*particleCount, hipMemcpyHostToDevice);
		reset = false;
	}
}

//ÿ֡����
extern "C" void runOnCuda()
{
	zeroGridCount << <cpu_gridBox.mDim.y, cpu_gridCount / cpu_gridBox.mDim.y >> >(gpu_orderedGridContainer);

	buidGridContent << <blockCount, particleCount / blockCount >> >(gpu_container, gpu_gridBox, gpu_orderedGridContainer);

	//�����ܶȺ�ѹ��
	computeDensityAndPressure << <blockCount, particleCount / blockCount >> >(gpu_orderedGridContainer, gpu_container, gpu_gridBox, gpu_poly6, gpu_neighborTable);

	//������
	computeForce << <blockCount, particleCount / blockCount >> >(gpu_container, gpu_kernelSpiky, gpu_kernelViscosity, gpu_neighborTable);

	//�ƶ�
	advance << <blockCount, particleCount / blockCount >> >(gpu_container, gpu_waterBox);

	if (boxIsChange)
	{
		updateBox();
	}

	//hipMemcpy(cpu_container, gpu_container, sizeof(Particle)*particleCount, hipMemcpyDeviceToHost);

}

extern "C" MyVector3::vector3 getParticlePos(int i)
{
	MyVector3::vector3 pos;
	void* tempP = &cpu_container[i].mPos;
	memcpy(&pos, tempP, sizeof(pos));
	return pos;
}

extern "C" int getParticleCount()
{
	return particleCount;
}

extern "C" void cudaSetD3DDevice()
{
	hipSetDevice(0);
}

extern "C" void registerResource(ID3D11Buffer* positionsVB)
{
	cudaGraphicsD3D11RegisterResource(&positionsVB_CUDA, positionsVB, hipGraphicsRegisterFlagsNone);
	cudaGraphicsResourceSetMapFlags(positionsVB_CUDA,cudaGraphicsMapFlagsWriteDiscard);
}

extern "C" void releaseCudaVB()
{
	hipGraphicsUnregisterResource(positionsVB_CUDA);
}

__global__ void writeInstanceData(MyVector3::vector3* positions, Particle*const container)
{
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x + ((blockIdx.y*blockDim.y) + threadIdx.y)*blockDim.x*gridDim.x;

	positions[tid].x = container[tid].mPos.x;
	positions[tid].y = container[tid].mPos.y;
	positions[tid].z = container[tid].mPos.z;
}

extern "C" void writeData()
{
	MyVector3::vector3* positions; 
	// mapʵ������
	hipGraphicsMapResources(1, &positionsVB_CUDA, 0); 
	size_t num_bytes; 
	hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes, positionsVB_CUDA); 
	// ִ��kernel
	writeInstanceData << <blockCount, particleCount / blockCount >> >(positions, gpu_container);
	// Unmap ʵ������
	hipGraphicsUnmapResources(1, &positionsVB_CUDA, 0);
}

float step = 0.1f;
extern "C" void changeWaterbox(int i)
{
	boxIsChange = true;
	switch (i)
	{
	case 0:
		waterBox.minX -= step; break;
	case 1:
		waterBox.minX += step; break;
	case 2:
		waterBox.maxX -= step; break;
	case 3:
		waterBox.maxX += step; break;
	case 4:
		reset = true; break;
	default:
		break;
	}
}